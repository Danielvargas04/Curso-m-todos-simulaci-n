
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
using namespace std;

#define Lx 31
#define Nx 8
const int Mx=(Lx+Nx-1)/Nx;

//--------------- KERNELS ----------------
__global__ void ReduceThreads(float * d_a,float * d_ReduceResult){
  //Set global index
  int ix=threadIdx.x+blockIdx.x*blockDim.x;
  //Define and load shared memory
  extern __shared__ unsigned int temp[];
  temp[threadIdx.x]=d_a[ix];
  __syncthreads();
  //Reduce loop
  for(unsigned int s=blockDim.x/2;s>0;s>>=1){
    if(threadIdx.x<s)
      temp[threadIdx.x]+=temp[threadIdx.x+s];
    __syncthreads();
  }
  //Write to d_ReduceResult
  if(threadIdx.x==0)
  {
    d_ReduceResult[blockIdx.x]=temp[0];
  }
}
int main(){
  //DECLARAR LAS MATRICES
  int ix,bx;
  //DECLARAR LAS MATRICES
  //En el Host
  float h_a[Lx],h_ReduceResult[Mx];
  //En el Device
  float *d_a;             hipMalloc((void**) &d_a,Lx*sizeof(float));
  float *d_ReduceResult;  hipMalloc((void**) &d_ReduceResult,Mx*sizeof(float));

  //INICIALIZAR LOS DATOS
  //Cargarlos en el Host
  for(ix=0;ix<Lx;ix++) h_a[ix]=ix;
  for(bx=0;bx<Mx;bx++) h_ReduceResult[bx]=0;
  //Enviarlos al Device
  hipMemcpy(d_a,h_a,Lx*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_ReduceResult,h_ReduceResult,Mx*sizeof(float),hipMemcpyHostToDevice);
  //Imprimirlos
  for(ix=0;ix<Lx;ix++) cout<<h_a[ix]<<" "; cout<<endl;
  //PROCESAR EN LA TARJETA GRAFICA
  dim3 ThreadsPerBlock(Nx,1,1);
  dim3 BlocksPerGrid(Mx,1,1);
  ReduceThreads<<<BlocksPerGrid,ThreadsPerBlock,Mx*sizeof(float)>>>(d_a,d_ReduceResult);

  //IMPRIMIR LOS DATOS
  //Devolverlos al Host
  hipMemcpy(h_ReduceResult,d_ReduceResult,Mx*sizeof(float),hipMemcpyDeviceToHost);
  //Imprimirlos
  for(bx=0;bx<Mx;bx++) cout<<h_ReduceResult[bx]<<" "; cout<<endl;

  //LIBERAR MEMORIA
  hipFree(d_a);  hipFree(d_ReduceResult);

  return 0;
}