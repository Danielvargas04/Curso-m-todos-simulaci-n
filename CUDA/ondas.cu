
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>

//Constantes del problemas

#define Lx 128
#define Ly 128
#define Q 5
#define N 32 //Threads per Block
const int M=(Lx*Ly+N-1)/N; //Blocks per Grid

const int ArraySize=Lx*Ly*Q;

const double W0=1.0/3;

const double C=0.5; //C<0.707 cells/click
const double C2=C*C;
const double AUX0=1-3*C2*(1-W0);

const double tau=0.5;
const double Utau=1.0/tau;
const double UmUtau=1-Utau;
//------ PROGRAMMING ON THE DIVICE (GPU)--------

__constant__ float d_w[5];
__constant__ int d_Vx[5];
__constant__ int d_Vy[5];

__constant__ float d_C[3]; //C[i]: C, C2, AUX0
__constant__ float d_tau[3]; //tau[i]: tau, Utau, UmUtau

//--------Funtions by device--------------

__device__ int d_n(int ix, int iy, int i)
{
    return (ix*Ly+iy)*Q+i;
}

__device__ float d_rho(int ix, int iy, float *d_f)
{
    float sum=0;
    int n0;
    for (int i = 0; i < Q; i++)
    {
        n0=d_n(ix,iy,i);
        sum+=d_f[n0];
    }
    return sum; 
}


__device__ float d_Jx(int ix, int iy, float *d_f)
{
    float sum=0;
    int n0;
    for (int i = 0; i < Q; i++)
    {
        n0=d_n(ix,iy,i);
        sum+=d_Vx[i]*d_f[n0];
    }
    return sum; 
}

__device__ float d_Jy(int ix, int iy, float *d_f)
{
    float sum=0;
    int n0;
    for (int i = 0; i < Q; i++)
    {
        n0=d_n(ix,iy,i);
        sum+=d_Vy[i]*d_f[n0];
    }
    return sum; 
}

__device__ float d_feq(float rho0, float Jx0, float Jy0, int i)
{
    return 3*d_w[i]*(d_C[1]*rho0+d_Vx[i]*Jx0+d_Vy[i]*Jy0);
}

__device__ float d_f0eq(float rho0, float Jx0, float Jy0)
{
    return rho0*d_C[2];
}

//-------------------------Kernels------------------------
__global__ void d_Collision(float *d_f, float *d_fnew)
{
    int ix, iy, i, n0, icell;
    float rho0, Jx0, Jy0;
    //Find which thread and which cell should I work
    icell=blockIdx.x*blockDim.x+threadIdx.x;
    ix=icell/Ly; iy=icell%Ly;
    //Compute the macroscopic fields
    rho0 = d_rho(ix, iy, d_f);
    Jx0 = d_Jx(ix, iy, d_f);
    Jy0 = d_Jy(ix, iy, d_f);
    //Collide and compute fnew
    n0=d_n(ix,iy,0);
    d_fnew[n0]=d_tau[2]*d_f[n0] + d_tau[1]*d_f0eq(rho0, Jx0, Jy0);
    for (i = 1; i < Q; i++)
    {
        n0=d_n(ix,iy,i);
        d_fnew[n0]=d_tau[2]*d_f[n0] + d_tau[1]*d_feq(rho0, Jx0, Jy0, i);
    }
}

__global__ void d_ImposeFields(float *d_f, float *d_fnew, float RhoSource)
{
    int ix, iy, i, n0;
    float rho0, Jx0, Jy0;
    //Find which thread and which cell should I work
    ix=Lx/2; iy=Ly/2;
    //Compute the macroscopic fields
    rho0 = RhoSource;
    Jx0 = d_Jx(ix, iy, d_f);
    Jy0 = d_Jy(ix, iy, d_f);
    //Collide and compute fnew
    n0=d_n(ix,iy,0);
    d_fnew[n0]=d_f0eq(rho0, Jx0, Jy0);
    for (i = 1; i < Q; i++)
    {
        n0=d_n(ix,iy,i);
        d_fnew[n0]=d_feq(rho0, Jx0, Jy0, i);
    }
}

__global__ void d_Advection(float *d_f, float *d_fnew)
{
    int ix, iy, i, n0, icell, ixnew, iynew, n0new;
    //Find which thread and which cell should I work
    icell = blockIdx.x*blockDim.x+threadIdx.x;
    ix=icell/Ly; iy=icell%Ly;
    //Move the contents to the neighboring cells
    for (i = 0; i < Q; i++)
    {
        ixnew=(ix+d_Vx[i]+Lx)%Lx; iynew=(iy+d_Vy[i]+Ly)%Ly;
        n0=d_n(ix,iy,i); n0new=d_n(ixnew,iynew,i);
        d_f[n0new]=d_fnew[n0];
    }
}


//---------------class Laticce--------------
class LatticeBoltzmann
{
private:
    float h_C[3]; //C[i]: C, C2, AUX0
    float h_tau[3]; //tau[i]: tau, Utau, UmUtau
    float h_w[Q];           //Weights
    int h_Vx[Q], h_Vy[Q];    //Velocity vectors
    float *h_f, *h_fnew;    
    float *d_f, *d_fnew;    
public:
    LatticeBoltzmann(void);
    ~LatticeBoltzmann();
    int n(int ix, int iy, int i){return (ix*Ly+iy)*Q+i;};
    //------------ Campos macroscopicos-----------------
    float h_rho(int ix, int iy);
    //------------ Funciones de equilibrio-----------------
    float feq(float rho0, float Jx0, float Jy0, int i);
    //---------- Evolucion temporal----------------
    void Star(float rho0, float Jx0, float Jy0);
    void Collision(void);
    void ImposeFields(int t);
    void Adveccion(void);
    //---------- Funciones Globales----------------
    void Print(const  char * NameFile);
};

//-------------Implementacion de funciones-----------------

LatticeBoltzmann::LatticeBoltzmann(void)
{
    //Set constans
    h_C[0]=C; h_C[1]=C2; h_C[2]=AUX0;
    h_tau[0]=tau; h_tau[1]=Utau; h_tau[2]=UmUtau; 
    //Set the weights
    h_w[0]=W0; h_w[1]=h_w[2]=h_w[3]=h_w[4]=(1.0-W0)/4;
    //Set Velocity vectors
    h_Vx[0]=0; h_Vx[1]=1; h_Vx[2]=0; h_Vx[3]=-1; h_Vx[4]=0;
    h_Vy[0]=0; h_Vy[1]=0; h_Vy[2]=1; h_Vy[3]=0; h_Vy[4]=-1;
    //------ Send to the divice------
    hipMemcpyToSymbol(HIP_SYMBOL(d_w), h_w,    Q*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Vx), h_Vx,  Q*sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Vy), h_Vy,  Q*sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_C), h_C,    3*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_tau), h_tau,3*sizeof(float), 0, hipMemcpyHostToDevice);

    //Create the dynamic arrays
    h_f=new float [ArraySize]; h_fnew=new float [ArraySize];
    //Build the dynamic matrices on the device
    hipMalloc((void**) &d_f, ArraySize*sizeof(float));
    hipMalloc((void**) &d_fnew, ArraySize*sizeof(float));
    
}

LatticeBoltzmann::~LatticeBoltzmann()
{
    delete[] h_f; delete[] h_fnew;
    hipFree(d_f); hipFree(d_fnew);
}

float LatticeBoltzmann::h_rho(int ix, int iy)
{
    float sum=0.0;
    int n0;
    for (int i = 0; i < Q; i++)
    {
        n0=n(ix,iy,i);
        sum+=h_fnew[n0];

    }
    return sum;    
}

float LatticeBoltzmann::feq(float rho0, float Jx0, float Jy0, int i)
{
    if(i>0)
        return 3*h_w[i]*(C2*rho0 + h_Vx[i]*Jx0 + h_Vy[i]*Jy0);
    else
        return rho0*AUX0;
}

void LatticeBoltzmann::Star(float rho0, float Jx0, float Jy0)
{
    int ix, iy, i, n0;
    for(ix = 0; ix < Lx; ix++)      //for each cell
        for(iy = 0; iy < Ly; iy++)
            for(i = 0; i < Q; i++)  //on each direction
            {
                n0=n(ix,iy,i);
                h_f[n0]=feq(rho0, Jx0, Jy0, i);
            }
    //send to the device
    hipMemcpy(d_f, h_f, ArraySize*sizeof(float), hipMemcpyHostToDevice);
}

void LatticeBoltzmann::Collision(void)
{
    //Do by device

    dim3 ThreadsPerBlock(N,1,1);
    dim3 BlocksPerGrid(M,1,1);
    d_Collision<<<BlocksPerGrid, ThreadsPerBlock>>>(d_f, d_fnew);
}

void LatticeBoltzmann::ImposeFields(int t)
{
    //Do by device  
    float lambda=10, omega=2*M_PI/lambda*C;
    float RhoSource=10*sin(omega*t);
    dim3 ThreadsPerBlock(1,1,1);
    dim3 BlocksPerGrid(1,1,1);
    d_ImposeFields<<<BlocksPerGrid, ThreadsPerBlock>>>(d_f, d_fnew, RhoSource);
}

void LatticeBoltzmann::Adveccion(void)
{
    //Do by device

    dim3 ThreadsPerBlock(N,1,1);
    dim3 BlocksPerGrid(M,1,1);
    d_Advection<<<BlocksPerGrid, ThreadsPerBlock>>>(d_f, d_fnew);
}

//--------------Funciones Globales--------------

void LatticeBoltzmann::Print(const char * NameFile)
{
    std::ofstream MyFile(NameFile); double rho0; int ix,iy;
    //Bring back the data from device to host
    hipMemcpy(h_fnew, d_fnew, ArraySize*sizeof(float), hipMemcpyDeviceToHost);
    for (ix = 0; ix < Lx; ix++)
    {
        for (iy = 0; iy < Ly; iy++)
        {
            rho0=h_rho(ix, iy);
            MyFile<<ix<<" "<<iy<<" "<<rho0<<std::endl;
        }
        MyFile<<std::endl;
    }
    MyFile.close();
}



int main(void)
{
    LatticeBoltzmann Ondas;
    int t, tmax=100;
    double rho0=0, Jx0=0, Jy0=0;

    //Start
    Ondas.Star(rho0, Jx0, Jy0);
    //Run
    for  (t = 0; t < tmax; t++)
    {
        Ondas.Collision();
        Ondas.ImposeFields(t);
        Ondas.Adveccion();
    }
    //show
    Ondas.Print("ondas.dat");

    return 0;
}


