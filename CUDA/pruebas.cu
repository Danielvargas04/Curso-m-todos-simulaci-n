
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>


#define Lx 16
#define Nx 8

const int Mx = (Lx+Nx-1)/Nx;

//--------------------KERNELS----------------
__global__ void AddTwoVectors(float *d_a,float *d_b,float *d_c){
 //Which thread should I do?
  int ix;  ix=blockIdx.x*blockDim.x+threadIdx.x;
  d_c[ix]=d_a[ix]+d_b[ix];}

int main()
{
    //DECLARE
    //Declare arrays in the Host
    float h_a[Lx],h_b[Lx],h_c[Lx];
    //Declare arrays in the Device
    float*d_a; hipMalloc((void**) &d_a,Lx*sizeof(float));
    float*d_b; hipMalloc((void**) &d_b,Lx*sizeof(float));
    float*d_c; hipMalloc((void**) &d_c,Lx*sizeof(float));

    //INPUT DATA
    //Set data in the Host
    for(int ix=0;ix<Lx;ix++)
    {
        h_a[ix]=ix; h_b[ix]=2*ix;
    }

    //Send data to the Device
    hipMemcpy(d_a,h_a,Lx*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,Lx*sizeof(float),hipMemcpyHostToDevice);

    //PROCESS
    //Run parallel on the Device
    dim3 ThreadsPerBlock(Nx,1,1);
    dim3 BlocksPerGrid(Mx,1,1);
    AddTwoVectors<<<BlocksPerGrid,ThreadsPerBlock>>>(d_a,d_b,d_c);

    //SHOW RESULTS
    //Bring back to the Host
    hipMemcpy(h_c,d_c,Lx*sizeof(float),hipMemcpyDeviceToHost);
    for(int ix=0;ix<Lx;ix++)
        std::cout<<ix<<" "<<h_c[ix]<<std::endl;

    //Free dynamic memory
    hipFree(d_a);  hipFree(d_b);  hipFree(d_c);
}